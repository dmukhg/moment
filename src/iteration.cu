#include "hip/hip_runtime.h"
#include "iteration.cuh"

/* Increment the time-step by 1.  
 * For use only with a single block and a single thread */
__global__ void time_step(int *dev_time)
{
    *dev_time += 1;
}

/* For all neurons with potential > IzTHRESHOLD, add their indices to
 * the fired array. Also reset the neurons which are firing. 
 *
 * Also, update firing rate of all neurons. */
__global__ void find_firing_neurons(Neuron *neurons, int *dev_time,
    bool *fired, int *rate, bool *fired_queue, int number)
{
  int offset = blockIdx.x * blockDim.x * blockDim.y + threadIdx.x;
  int queue_offset = *dev_time % FIRED_RES;

  float v, u;

  if (offset >= number) {
    // There are no such neurons
    return;
  }

  v = neurons[offset].potential;
  u = neurons[offset].recovery;

  if (v > IzTHRESHOLD) {
    fired[offset] = true;

    // reset firing neuruons
    neurons[offset].potential = IzC;
    neurons[offset].recovery  = u + IzD;

    // Add this firing to the queue
    // Compute spike firing rate
    if (!fired_queue[FIRED_RES * offset + queue_offset]) {
      // If there wasn't a spike at this offset, increment spike rate.
      // If there was, doesn't really matter.
      rate[offset] += 1;
    }

    fired_queue[FIRED_RES * offset + queue_offset] = true;
  } else {
    // Spike rate adjustments

    // If there was a spike at this offset, decrement spike rate.
    // If there wasn't doesn't matter
    if (fired_queue[FIRED_RES * offset + queue_offset]) {
      rate[offset] -= 1;
      fired_queue[FIRED_RES * offset + queue_offset] = false;
    }
  }
}

/* For all fired neurons, update the thalamic input on connected
 * neurons.*/
__global__ void update_current(Neuron *neurons, Connection *connections,
    bool *fired, int number) {
  int offset = blockIdx.x * blockDim.x * blockDim.y + threadIdx.x;
  int cIdx, nIdx;

  // First, make the current on all neurons 0 + the input value for
  // that neuron
  if (offset < number) {
    neurons[offset].current = 0 + 1000*neurons[offset].input;
  }
  // Ensure that *all* neurons have 0 current
  __syncthreads();

  if (fired == NULL || fired[offset] == false) {
    // No such fired neuron
    return;
  }

  cIdx    = neurons[offset].connection;

  if (connections != NULL) {
    do {
      nIdx = connections[cIdx].neuron;
      atomicAdd(&(neurons[nIdx].current), 1000*connections[cIdx].weight);
     // atomicAdd(&neurons[connections[cIdx].neuron].current,
      //    1.0f);
      cIdx = connections[cIdx].next;
    } while (cIdx >= 0);
  }
}
