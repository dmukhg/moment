#include "hip/hip_runtime.h"
#include <stdio.h>

#include "main.cuh"

/* Step forward in time. */
__global__ void time_step(int *dev_time)
{
  *dev_time += 1; 
}

__global__ void change_conn(Connection *dev_connections) 
{
  dev_connections[dev_get_index(1,12)].neuron = 14141;
}

int main( void ) 
{
    int *dev_time;
    int host_time = 0;

    Connection host_connections[NUMNEURON * PSYNCONN]; 
    Connection *dev_connections;

    // Allocate memory on the GPU
    hipMalloc( (void**)&dev_time, sizeof(int) );
    hipMalloc( (void**)&dev_connections,
        sizeof(Connection) * NUMNEURON * PSYNCONN);

    // Copy the time to the GPU
    hipMemcpy(dev_time, &host_time, sizeof(int),
        hipMemcpyHostToDevice);

    // Copy the connections to the GPU
    hipMemcpy(dev_connections, &host_connections,
        sizeof(Connection) * NUMNEURON * PSYNCONN,
        hipMemcpyHostToDevice);

    // XXX The limit shouldn't be iterations. Discuss!
    while (host_time < ITERATIONS) {
        // Step forward in time.  Since this is a part of the global
        // memory, you only need to do it via one thread.
        time_step<<<1,1>>>(dev_time);

        // copy the time back to the cpu
        hipMemcpy(&host_time, dev_time, sizeof(int),
            hipMemcpyDeviceToHost); 
    }

    change_conn<<<1,1>>>(dev_connections);

    hipMemcpy(&host_connections, dev_connections,
        sizeof(Connection) * NUMNEURON * PSYNCONN,
        hipMemcpyDeviceToHost);

    printf("%d\n", host_connections[get_index(1,12)].neuron);

    // Free memory on GPU
    hipFree(dev_time);

    return 0; 
}
