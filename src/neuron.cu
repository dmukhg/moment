#include "hip/hip_runtime.h"
#include "neuron.cuh"

/* Establishes a random value of synaptic current on the input
 * neurons. */
void input_random_current(Neuron *neurons) 
{
  int i, r;

  for (i=0; i < INNEURON; i++) {
    r = rand() % 10; // Assumes seeding has been done

    neurons[i].current = (r > 5) ? 5 : 0.0;
  }
}

/* Go through each neuron and update the membrane potential based on
 * the Izhikewich model. This is for a single iteration. */
__global__ void update_potential(Neuron *neurons, int number)
{
  float del_v, del_u, v, u, I;
              
  int offset = blockIdx.x * blockDim.x * blockDim.y + threadIdx.x;

  if (offset >= number) { 
    // There are no such neurons
    return;
  }

  v = neurons[offset].potential;
  u = neurons[offset].recovery;
  I = neurons[offset].current;

  if (v > IzTHRESHOLD) {
    neurons[offset].potential = IzC;
    neurons[offset].recovery = u + IzD;
  } else {
    del_v = 0.04f*v*v + 5.0f*v + 140.0f - u + I;
    del_u = IzA * ( IzB*v - u);

    neurons[offset].potential = v + del_v;
    neurons[offset].recovery  = u + del_u;
  }
}
