/* Test neuron functionality */
#define NEU_COUNT 1

#include <stdio.h>

// Note that we are including the source file and not the header file only.
// This allows us access to static variables and globals (gasp!). This is more
// or less a close replica of the exposure that the functions being tested will
// get.
#include "neuron.cu"
#include "iteration.cu"

/* Study the spike pattern in a single neuron with a stepped input. 
 * The output of this program is meant to be viewed with the included spiking
 * visualizer in test/visualizer/. */
int test_single_neuron(void)
{
  int *dev_time;
  int host_time = 0;
  bool *dev_fired, host_fired[1];

  Neuron host_neurons[1];
  Neuron *dev_neurons;

  int host_rate[NEU_COUNT];
  int *dev_rate;

  bool host_fired_queue[NEU_COUNT * FIRED_RES], *dev_fired_queue;

  // Allocate memory on the GPU
  hipMalloc( (void**)&dev_time, sizeof(int));
  hipMalloc( (void**)&dev_neurons, sizeof(Neuron));
  hipMalloc( (void**)&dev_rate, sizeof(int) * NEU_COUNT);
  hipMalloc( (void**)&dev_fired_queue, sizeof(bool) * FIRED_RES * NEU_COUNT);

  // Initialization
  host_neurons[0].current = 0;
  fill_false(host_fired, 1);
  fill_false(host_fired_queue, NEU_COUNT * FIRED_RES);
  fill_zeros(host_rate, NEU_COUNT);
  hipMemcpy(dev_time, &host_time, sizeof(int),
      hipMemcpyHostToDevice);
  hipMemcpy(dev_neurons, &host_neurons, sizeof(Neuron),
      hipMemcpyHostToDevice);
  hipMemcpy(dev_fired_queue, &host_fired_queue, sizeof(bool) * NEU_COUNT * FIRED_RES,
      hipMemcpyHostToDevice);
  hipMemcpy(dev_rate, &host_rate, sizeof(int) * NEU_COUNT,
      hipMemcpyHostToDevice);

  while (host_time < 1000) {
    time_step<<<1,1>>>(dev_time);
    hipMalloc( (void**)&dev_fired, sizeof(bool));
    hipMemcpy(dev_fired, &host_fired, sizeof(bool), hipMemcpyHostToDevice);

    if (host_time == 500) {
      // At t=400, give thalamic input of 4 to the neuron
      host_neurons[0].input = 13;
      hipMemcpy(dev_neurons, host_neurons, sizeof(Neuron),
          hipMemcpyHostToDevice);
    }

    if (host_time == 800) {
      // At t=800, remove all thalamic input
      host_neurons[0].input = 0.0f;
      hipMemcpy(dev_neurons, host_neurons, sizeof(Neuron),
          hipMemcpyHostToDevice);
    }

    find_firing_neurons<<<1,1>>>(dev_neurons, dev_time, dev_fired, dev_rate,
        dev_fired_queue, 1);
    update_current<<<1,1>>>(dev_neurons, NULL, dev_fired, 1);
    update_potential<<<1,1>>>(dev_neurons, NULL, 1);

    hipMemcpy(host_neurons, dev_neurons, sizeof(Neuron),
        hipMemcpyDeviceToHost);

    printf("[ %d, %10f],\n", host_time, host_neurons[0].potential);

    host_time++;
  }

  return 0;
}


int main(void) 
{
  test_single_neuron();

  return 0;
}
