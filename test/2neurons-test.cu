/* Test network functionality */
#define B2S(c) ((c) ? 't': 'f') 
#define NEU_COUNT 2

#include <stdio.h>

// Note that we are including the source file and not the header file only.
// This allows us access to static variables and globals (gasp!). This is more
// or less a close replica of the exposure that the functions being tested will
// get.
#include "neuron.cu"
#include "iteration.cu"

int test_two_neuron_network(void)
{
  int *dev_time;
  int host_time = 0;

  Neuron host_neurons[2];
  Neuron *dev_neurons;

  int host_rate[NEU_COUNT];
  int *dev_rate;

  Connection host_connections[1];
  Connection *dev_connections;

  bool *dev_fired, host_fired[2];
  bool host_fired_queue[NEU_COUNT * FIRED_RES], *dev_fired_queue;

  // Allocate memory on the GPU
  hipMalloc( (void**)&dev_time, sizeof(int));
  hipMalloc( (void**)&dev_neurons, sizeof(Neuron)*2);
  hipMalloc( (void**)&dev_connections, sizeof(Connection));
  hipMalloc( (void**)&dev_rate, sizeof(int) * NEU_COUNT);
  hipMalloc( (void**)&dev_fired_queue, sizeof(bool) * FIRED_RES * NEU_COUNT);

  // Initialization
  fill_false(host_fired, 2);
  fill_false(host_fired_queue, NEU_COUNT * FIRED_RES);
  fill_zeros(host_rate, NEU_COUNT);
  host_neurons[0].connection = 0;
  host_connections[0].next = -1;
  host_connections[0].weight = 20.0;
  host_connections[0].neuron = 1; 

  // Copy all to device
  hipMemcpy(dev_time, &host_time, sizeof(int),
      hipMemcpyHostToDevice);
  hipMemcpy(dev_neurons, &host_neurons, sizeof(Neuron)*2,
      hipMemcpyHostToDevice);
  hipMemcpy(dev_connections, &host_connections, sizeof(Connection),
      hipMemcpyHostToDevice);
  hipMemcpy(dev_fired_queue, &host_fired_queue, sizeof(bool) * NEU_COUNT * FIRED_RES,
      hipMemcpyHostToDevice);
  hipMemcpy(dev_rate, &host_rate, sizeof(int) * NEU_COUNT,
      hipMemcpyHostToDevice);


  while (host_time < 1000) {
    time_step<<<1,1>>>(dev_time);
    hipMalloc( (void**)&dev_fired, sizeof(bool)*2);
    hipMemcpy(dev_fired, &host_fired, sizeof(bool)*2, hipMemcpyHostToDevice);

    if (host_time == 500) {
      // At t=500, give thalamic input of 4 to the neuron
      host_neurons[0].input = 4;
      hipMemcpy(dev_neurons, host_neurons, sizeof(Neuron)*2,
          hipMemcpyHostToDevice);
    }

    find_firing_neurons<<<1,NEU_COUNT>>>(dev_neurons, dev_time, dev_fired, dev_rate,
        dev_fired_queue, NEU_COUNT);
    update_current<<<1,2>>>(dev_neurons, dev_connections, dev_fired, 2);
    update_potential<<<1,2>>>(dev_neurons, dev_connections, 2);

    hipMemcpy(host_neurons, dev_neurons, sizeof(Neuron)*2,
        hipMemcpyDeviceToHost);

    //printf("[ %d, %10d],\n", host_time, host_neurons[1].current);
    printf("[ %d, %10f],\n", host_time, host_neurons[1].potential);

    hipFree(&dev_fired);
    host_time++;
  }

  return 0;
}

int main() {
  test_two_neuron_network();

  return 0;
}
