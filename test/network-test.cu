/* Test network functionality */

#include <stdio.h>

// Note that we are including the source file and not the header file only.
// This allows us access to static variables and globals (gasp!). This is more
// or less a close replica of the exposure that the functions being tested will
// get.
#include "neuron.cu"
#include "iteration.cu"

int test_two_neuron_network(void)
{
  int *dev_time;
  int host_time = 0;

  Neuron host_neurons[2];
  Neuron *dev_neurons;

  Connection host_connections[1];
  Connection *dev_connections;

  // Allocate memory on the GPU
  hipMalloc( (void**)&dev_time, sizeof(int));
  hipMalloc( (void**)&dev_neurons, sizeof(Neuron)*2);
  hipMalloc( (void**)&dev_connections, sizeof(Connection));

  // Initialization
  host_neurons[0].current = 0;
  host_neurons[0].connection = 0;
  host_connections[0].next = 0;
  host_connections[0].weight = 0.1;
  host_connections[0].neuron = 1; 

  // Copy all to device
  hipMemcpy(dev_time, &host_time, sizeof(int),
      hipMemcpyHostToDevice);
  hipMemcpy(dev_neurons, &host_neurons, sizeof(Neuron)*2,
      hipMemcpyHostToDevice);
  hipMemcpy(dev_connections, &host_connections, sizeof(Connection),
      hipMemcpyHostToDevice);

  while (host_time < 1000) {
    time_step<<<1,1>>>(dev_time);

    if (host_time == 500) {
      // At t=500, give thalamic input of 4 to the neuron
      host_neurons[0].current = 4.0f;
      hipMemcpy(dev_neurons, host_neurons, sizeof(Neuron)*2,
          hipMemcpyHostToDevice);
    }

    update_potential<<<1,2>>>(dev_neurons, dev_connections, 2);

    hipMemcpy(host_neurons, dev_neurons, sizeof(Neuron)*2,
        hipMemcpyDeviceToHost);

    printf("[ %d, %10f],\n", host_time, host_neurons[1].potential);

    host_time++;
  }

  return 0;
}

int main() {
  test_two_neuron_network();

  return 0;
}
